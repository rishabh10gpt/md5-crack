#include "hip/hip_runtime.h"
#include <unistd.h>
#include <getopt.h>
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <string>
#include <cstring>
#include <cctype>
#include <cstdint>
#include <sstream>
#include <mpi.h>
#include<ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>


#define ERROR_CHECK(X) { gpuAssert((X), __FILE__, __LINE__); }

#define CONST_WORD_LIMIT 20
#define CONST_CHARSET_LIMIT 100

#define CONST_WORD_LENGTH_MIN 10
#define CONST_WORD_LENGTH_MAX 16

#define TOTAL_BLOCKS 16384UL
#define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 128UL

#include "md5.cu"

/* Global variables */
char CONST_CHARSET[CONST_CHARSET_LIMIT];
uint32_t CONST_CHARSET_LENGTH;

uint8_t g_wordLength;

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LIMIT];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];


struct arguments
{
    int charset;
    int min_len;
    int max_len;
    std::string hash;
};

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment, uint32_t CONST_CHARSET_LENGTH){
  uint32_t idx = 0;
  uint32_t add = 0;

  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }

    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }

  if(idx > *length){
    *length = idx;
  }

  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04, uint32_t CONST_CHARSET_LENGTH,uint64_t start_index, uint64_t end_index){
  uint32_t idx =  start_index + (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  if (idx >= end_index) return;
  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LIMIT];

  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx, CONST_CHARSET_LENGTH);

  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    if (idx >= end_index) break;

    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }

    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

    if(threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04){
      memcpy(g_deviceCracked, threadTextWord, threadWordLength);
    }

    if(!next(&threadWordLength, threadCharsetWord, 1, CONST_CHARSET_LENGTH)){
      break;
    }
    idx += 1;
  }
}

int main(int argc, char* argv[]){
  int opt = 0;
  struct arguments args;

  /* Default values. */
  args.charset = -1;
  args.hash = "";
  args.min_len = CONST_WORD_LENGTH_MIN;
  args.max_len = CONST_WORD_LENGTH_MAX;
  static struct option long_options[] = {
      {"charset", required_argument, NULL, 1},
      {"hash",    required_argument, NULL, 2},
      {"min_len", required_argument, NULL, 3},
      {"max_len", required_argument, NULL, 4},
      {NULL,      0,                 NULL, 0}
  };

    while ((opt = getopt_long(argc, argv, "", long_options, NULL)) != -1) {
      switch (opt) {
        case 1:
          args.charset = atoi(optarg);
          break;
        case 2:
          args.hash = optarg;
          break;
	case 3:
	  args.min_len = atoi(optarg);
	  break;
	case 4:
	  args.max_len = atoi(optarg);
	  break;
        default:
            std::cout << "Incorrect command line argument" << std::endl;
            exit(EXIT_FAILURE);
      }
    }
    std::cout<< "Parsed arguments: charset="<< args.charset<< ", hash="<<args.hash<< ", min_len="<<args.min_len<<", max_len="<<args.max_len<<std::endl;
    // Check mandatory parameters:
    if (args.hash.empty()) {
      std::cout << "--hash is mandatory" << std::endl;
      exit(EXIT_FAILURE);
    }

    // Check if hash if 32 hexadecimal character
    if (args.hash.length() != 32) {
      std::cout << "The hash must be 32 hexadecimal digits" << std::endl;
      exit(EXIT_FAILURE);
    }
    else {
      for (int i = 0; i < args.hash.length(); i++) {
        if (!isxdigit(args.hash[i])) {
          std::cout << "The hash must be 32 hexadecimal digits" << std::endl;
          exit(EXIT_FAILURE);
        }
      }
    }

    if (args.charset == 0) {
      strcpy(CONST_CHARSET, "0123456789");
      CONST_CHARSET_LENGTH = strlen(CONST_CHARSET);
    }
    else if (args.charset == 1) {
      strcpy(CONST_CHARSET, "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789");
      CONST_CHARSET_LENGTH = strlen(CONST_CHARSET);
    }
    else if (args.charset ==2) {
      strcpy(CONST_CHARSET, "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!#$%&'()*+,-./:;<=>?@[\\]^_`{|}~\"");
      CONST_CHARSET_LENGTH = strlen(CONST_CHARSET);
    }
    else {
      std::cout << "--charset is mandatory (0 for numeric or 1 for alphanumeric)" << std::endl;
      exit(EXIT_FAILURE);
    }
    // Initialize MPI
    MPI_Init(&argc, &argv);

    // Get MPI rank and size
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));

  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

  /* Display amount of devices */
  if (rank==0){
    std::cout << "Info: " << devices << " device(s) found" << std::endl;
  }


  /* Adding this logic to divide the jobs among all gpus at each node */
  uint64_t total_words = pow(CONST_CHARSET_LENGTH, CONST_WORD_LENGTH_MAX);
  //int start_device_index = rank * devices;
  //int end_device_index = start_device_index + devices;
  //Divide the words
  uint64_t total_iterations = TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS;
  uint64_t iterations_per_process = total_iterations / size;
  uint64_t start_index = rank * iterations_per_process;
  uint64_t end_index = (rank+1)* iterations_per_process -1; // (rank == size-1 ) ? total_iterations: start_index + iterations_per_process;
  uint64_t iterations_per_device = total_iterations / (size * devices);
  /* Hash stored as u32 integers */
  uint32_t md5Hash[4];
  const char *hash_cstr = args.hash.c_str();

  /* Parse argument */
  for(uint8_t i = 0; i < 4; i++){
    char tmp[16];

    strncpy(tmp, hash_cstr + i * 8, 8);
    sscanf(tmp, "%x", &md5Hash[i]);
    md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
  }

  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);

  /* Current word length = minimum word length */
  g_wordLength = args.min_len;

  /* Main device */
  hipSetDevice(0);

  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;

  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventRecord(clockBegin, 0);

  /* Current word is different on each device */
  char** words = new char*[devices];

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));

    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
  }
  
  while(true){
    bool result = false;
    bool found = false;

    for(int device = 0; device < devices; device++){
      hipSetDevice(device);
      
      // Calculate the global start index for each device
      uint64_t global_device_id = rank * devices + device;
      start_index = global_device_id * iterations_per_device;
      end_index = (global_device_id + 1) * iterations_per_device;
      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice));
      //std::cout << "rank: "<<rank<< ",  start index: "<< start_index << " , end index: "<< end_index << std::endl;
      /* Start kernel */
      md5Crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[device], md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3], CONST_CHARSET_LENGTH, start_index, end_index);

    }

    for(int device = 0; device < devices; device++){
      hipSetDevice(device);

      /* Synchronize now */
      hipDeviceSynchronize();

      /* Copy result */
      ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost));
	

      static uint8_t previousLength = 0;
      if(g_wordLength != previousLength && rank==0){
        time_t timenow = time(NULL); 
        std::cout << "Current word length: "<< (int)g_wordLength<< ", Current time: "<<ctime(&timenow)<< std::endl;
        
	previousLength = g_wordLength;
      }
      /* Check result */
      if(found = *g_cracked != 0){
        std::cout <<"Node : "<<  rank<<  " found the password for MD5 hash " << args.hash << " is " << g_cracked << std::endl;
        break;
      }
    }
    bool global_found = false;
    // Check if any node found the hash
    MPI_Allreduce(&found, &global_found, 1, MPI_CXX_BOOL, MPI_LOR, MPI_COMM_WORLD);
    /* Check result */
    if(global_found){
        break;
    }
    result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS, CONST_CHARSET_LENGTH); 
    if(!result){
        break;
    }
  } 
  // Finalize MPI
  MPI_Finalize();
  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Free on each device */
    hipFree((void**)words[device]);
  }

  /* Free array */
  delete[] words;

  /* Main device */
  hipSetDevice(0);

  float milliseconds = 0;

  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

  // std::cout << "Notice: computation time " << milliseconds << " ms" << std::endl;

  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);

  return 0;
}
