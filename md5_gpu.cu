#include "hip/hip_runtime.h"
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <unistd.h>
#include <getopt.h>
#include <cstdio>
#include <iostream>
#include <cstdlib>
#include <string>
#include <cstring>
#include <cctype>
#include <cstdint>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>


#define ERROR_CHECK(X) { gpuAssert((X), __FILE__, __LINE__); }

#define CONST_WORD_LIMIT 20
#define CONST_CHARSET_LIMIT 100

#define CONST_WORD_LENGTH_MIN 1
#define CONST_WORD_LENGTH_MAX 16

#define TOTAL_BLOCKS 16384UL
#define TOTAL_THREADS 512UL
#define HASHES_PER_KERNEL 128UL

#include "md5.cu"

/* Global variables */
char CONST_CHARSET[CONST_CHARSET_LIMIT];
uint32_t CONST_CHARSET_LENGTH;

uint8_t g_wordLength;

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LIMIT];
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LIMIT];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];


struct arguments
{
    int charset;
    std::string hash;
};

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment, uint32_t CONST_CHARSET_LENGTH){
  uint32_t idx = 0;
  uint32_t add = 0;

  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }

    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }

  if(idx > *length){
    *length = idx;
  }

  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04, uint32_t CONST_CHARSET_LENGTH){
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;

  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LIMIT];

  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx, CONST_CHARSET_LENGTH);

  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }

    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

    if(threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04){
      memcpy(g_deviceCracked, threadTextWord, threadWordLength);
    }

    if(!next(&threadWordLength, threadCharsetWord, 1, CONST_CHARSET_LENGTH)){
      break;
    }
  }
}

int main(int argc, char* argv[]){
  int opt = 0;
  struct arguments args;

  /* Default values. */
  args.charset = -1;
  args.hash = "";

  static struct option long_options[] = {
      {"charset", required_argument, NULL, 1},
      {"hash",    required_argument, NULL, 2},
      {NULL,      0,                 NULL, 0}
  };

    while ((opt = getopt_long(argc, argv, "", long_options, NULL)) != -1) {
      switch (opt) {
        case 1:
          args.charset = atoi(optarg);
          break;
        case 2:
          args.hash = optarg;
          break;
        default:
            std::cout << "Incorrect command line argument" << std::endl;
            exit(EXIT_FAILURE);
      }
    }

    // Check mandatory parameters:
    if (args.hash.empty()) {
      std::cout << "--hash is mandatory" << std::endl;
      exit(EXIT_FAILURE);
    }

    // Check if hash if 32 hexadecimal character
    if (args.hash.length() != 32) {
      std::cout << "The hash must be 32 hexadecimal digits" << std::endl;
      exit(EXIT_FAILURE);
    }
    else {
      for (int i = 0; i < args.hash.length(); i++) {
        if (!isxdigit(args.hash[i])) {
          std::cout << "The hash must be 32 hexadecimal digits" << std::endl;
          exit(EXIT_FAILURE);
        }
      }
    }

    if (args.charset == 0) {
      strcpy(CONST_CHARSET, "0123456789");
      CONST_CHARSET_LENGTH = strlen(CONST_CHARSET);
    }
    else if (args.charset == 1) {
      strcpy(CONST_CHARSET, "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789");
      CONST_CHARSET_LENGTH = strlen(CONST_CHARSET);
    }
    else {
      std::cout << "--charset is mandatory (0 for numeric or 1 for alphanumeric)" << std::endl;
      exit(EXIT_FAILURE);
    }

  /* Amount of available devices */
  int devices;
  ERROR_CHECK(hipGetDeviceCount(&devices));

  /* Sync type */
  ERROR_CHECK(hipSetDeviceFlags(hipDeviceScheduleSpin));

  /* Display amount of devices */
  std::cout << "Info: " << devices << " device(s) found" << std::endl;

  /* Hash stored as u32 integers */
  uint32_t md5Hash[4];
  const char *hash_cstr = args.hash.c_str();

  /* Parse argument */
  for(uint8_t i = 0; i < 4; i++){
    char tmp[16];

    strncpy(tmp, hash_cstr + i * 8, 8);
    sscanf(tmp, "%x", &md5Hash[i]);
    md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
  }

  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);

  /* Current word length = minimum word length */
  g_wordLength = CONST_WORD_LENGTH_MIN;

  /* Main device */
  hipSetDevice(0);

  /* Time */
  hipEvent_t clockBegin;
  hipEvent_t clockLast;

  hipEventCreate(&clockBegin);
  hipEventCreate(&clockLast);
  hipEventRecord(clockBegin, 0);

  /* Current word is different on each device */
  char** words = new char*[devices];

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Copy to each device */
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));

    /* Allocate on each device */
    ERROR_CHECK(hipMalloc((void**)&words[device], sizeof(uint8_t) * CONST_WORD_LIMIT));
  }

  while(true){
    bool result = false;
    bool found = false;

    for(int device = 0; device < devices; device++){
      hipSetDevice(device);

      /* Copy current data */
      ERROR_CHECK(hipMemcpy(words[device], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice));

      /* Start kernel */
      md5Crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[device], md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3], CONST_CHARSET_LENGTH);

      /* Global increment */
      result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS, CONST_CHARSET_LENGTH);
    }

    /* Display progress */
    // char word[CONST_WORD_LIMIT];

    // for(int i = 0; i < g_wordLength; i++){
    //   word[i] = g_charset[g_word[i]];
    // }

    // std::cout << "Notice: currently at " << std::string(word, g_wordLength) << " (" << (uint32_t)g_wordLength << ")" << std::endl;

    for(int device = 0; device < devices; device++){
      hipSetDevice(device);

      /* Synchronize now */
      hipDeviceSynchronize();

      /* Copy result */
      ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost));

      /* Check result */
      if(found = *g_cracked != 0){
        std::cout << "The password for MD5 hash " << args.hash << " is " << g_cracked << std::endl;
        break;
      }
    }

    if(!result || found){
      if(!result && !found){
        std::cout << "Notice: found nothing (host)" << std::endl;
      }

      break;
    }
  }

  for(int device = 0; device < devices; device++){
    hipSetDevice(device);

    /* Free on each device */
    hipFree((void**)words[device]);
  }

  /* Free array */
  delete[] words;

  /* Main device */
  hipSetDevice(0);

  float milliseconds = 0;

  hipEventRecord(clockLast, 0);
  hipEventSynchronize(clockLast);
  hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

  // std::cout << "Notice: computation time " << milliseconds << " ms" << std::endl;

  hipEventDestroy(clockBegin);
  hipEventDestroy(clockLast);

  return 0;
}
